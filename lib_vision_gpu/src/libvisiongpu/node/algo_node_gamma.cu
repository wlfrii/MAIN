#include "hip/hip_runtime.h"
#ifndef ALGONODEGAMMA_CU
#define ALGONODEGAMMA_CU
#include "algo_node_gamma.h"
#include "../def/cu_define.h"
#include "../func/func_colorspace.h"

namespace
{
	/* img_hsv = rgb2hsv(src);
	 * y = img_hsv(:, :, 3);
	 * filtered_y = ipo.guidedFilter(y, y, 4, 0.01);
	 * m = mean2(filtered_y); 
	 * gamma = m. ^ ((m - filtered_y). / m);
	 * y_new = y.^gamma;
	 * img_hsv(:, :, 3) = y_new;
	 * dst = hsv2rgb(img_hsv);*/
    __global__ void calcGammaTransform(cv::cuda::PtrStepSz<float3> hsv, cv::cuda::PtrStepSz<float> filtered_y, float mean_y/*, cv::cuda::PtrStepSz<float> ga*/)
    {
        int thread_id = _get_threadId_grid2D_block1D();
		int row = thread_id / filtered_y.cols;
		int col = thread_id % filtered_y.cols;
		if (row < filtered_y.rows && col < filtered_y.cols)
		{
			float gamma = powf(mean_y, 1.0 - filtered_y(row, col) / mean_y);
			/*ga(row, col) = gamma;*/
			float y_new = powf(hsv(row, col).z, gamma);

			hsv(row, col).z = MAX(MIN(y_new, 1), 0);
		}
    }

	__global__ void calcGammaTransform(cv::cuda::PtrStepSz<float3> hsv, cv::cuda::PtrStepSz<float> filtered_y, float alpha, float ref_L)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int row = thread_id / filtered_y.cols;
		int col = thread_id % filtered_y.cols;
		if (row < filtered_y.rows && col < filtered_y.cols)
		{
			float gamma = powf(alpha, 1.0 - filtered_y(row, col) / ref_L);
			float y_new = powf(hsv(row, col).z, gamma);

			hsv(row, col).z = MAX(MIN(y_new, 1), 0);
		}
	}
}

GPU_ALGO_BEGIN
void AdaptiveGamma(cv::cuda::GpuMat &src, cv::cuda::GpuMat &v, hipStream_t &stream, cv::cuda::GpuMat &tmp, float alpha, float ref_L)
{	
	gpu::cvtColor(src, tmp, BGRA2HSV);
#if CU_DEBUG
	cv::Mat test_hsv; tmp.download(test_hsv);
#endif

	if (abs(alpha) < 0.05 && abs(ref_L) < 0.05) {
		cv::Scalar mean_rgba = cv::cuda::sum(v);
		float mean_v = mean_rgba(0) / (v.rows * v.cols);

		::calcGammaTransform << < dim3(90, 90), 256, 0, stream >> > (tmp, v, mean_v);
	}
	else {
		::calcGammaTransform << < dim3(90, 90), 256, 0, stream >> > (tmp, v, alpha, ref_L);
	}

#if CU_DEBUG
	cv::Mat test_tmp; tmp.download(test_tmp);
#endif

	gpu::cvtColor(tmp, src, HSV2BGRA);

#if CU_DEBUG
	cv::Mat test_res; src.download(test_res);
#endif
}

GPU_ALGO_END
#endif //ALGONODEGAMMA_CU
