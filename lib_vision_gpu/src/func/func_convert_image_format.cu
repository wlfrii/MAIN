#include "hip/hip_runtime.h"
#include "../gpu_algorithm_func.h"
#include "../def/cu_define.h"

namespace
{
	__global__ void cvt8UC1to32FC1(cv::cuda::PtrStepSz<uchar> src, cv::cuda::PtrStepSz<float> cvt_src)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int col = thread_id % src.cols;
		int row = thread_id / src.cols;

		if (col < src.cols && row < src.rows)
		{
			cvt_src(row, col) = float(src(row, col)) / 255.f;
		}
	}
	__global__ void cvt8UC3to32FC3(cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSz<float3> cvt_src)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int col = thread_id % src.cols;
		int row = thread_id / src.cols;

		if (col < src.cols && row < src.rows)
		{
			cvt_src(row, col).x = float(src(row, col).x) / 255.f;
			cvt_src(row, col).y = float(src(row, col).y) / 255.f;
			cvt_src(row, col).z = float(src(row, col).z) / 255.f;
		}
	}
	__global__ void cvt8UC4to32FC4(cv::cuda::PtrStepSz<uchar4> src, cv::cuda::PtrStepSz<float4> cvt_src)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int col = thread_id % src.cols;
		int row = thread_id / src.cols;

		if (col < src.cols && row < src.rows)
		{
			cvt_src(row, col).x = float(src(row, col).x) / 255.f;
			cvt_src(row, col).y = float(src(row, col).y) / 255.f;
			cvt_src(row, col).z = float(src(row, col).z) / 255.f;
			cvt_src(row, col).w = float(src(row, col).w) / 255.f;
		}
	}
}

GPU_ALGO_BEGIN
void convertImageFormat(cv::cuda::GpuMat &src, hipStream_t stream)
{
#if CU_DEBUG
	//cv::Mat test_src; src.download(test_src);
#endif

	auto imfmt = src.type();
	if (imfmt == CV_8UC1) {
		cv::cuda::GpuMat tmp1(src.size(), CV_32FC1);
		::cvt8UC1to32FC1 << < dim3(90, 90), 256, 0, stream >> > (src, tmp1);
		src = tmp1;
	}
	else if (imfmt == CV_8UC3) {
		cv::cuda::GpuMat tmp3(src.size(), CV_32FC3);
		::cvt8UC3to32FC3 << < dim3(90, 90), 256, 0, stream >> > (src, tmp3);
		src = tmp3;
	}
	else if (imfmt == CV_8UC4) {
		cv::cuda::GpuMat tmp4(src.size(), CV_32FC4);
		::cvt8UC4to32FC4 << < dim3(90, 90), 256, 0, stream >> > (src, tmp4);
		src = tmp4;
	}
#if CU_DEBUG
	//cv::Mat test_res; src.download(test_res);
#endif
}

GPU_ALGO_END