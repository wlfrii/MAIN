#include "hip/hip_runtime.h"
#include "algo_node_uneven_y.h"
#include "../def/cu_define.h"


namespace
{
	/*function uneven_y = lumalaws(width, height, distance, I)
		% calculate the distance between pixel point and image center
		[X, Y] = meshgrid(1:width, 1 : height);
		radius_square = (X - floor(width / 2)). ^ 2 + (Y - floor(height / 2)). ^ 2;
		% calculate the angle
		theta = atan2(sqrt(radius_square), distance);
		% calculate L
		L_square = radius_square + distance ^ 2;
		% calculate intensity of pixel
		uneven_y = I * cos(theta). / L_square;

		% remap the y_image
		max_y = max(max(uneven_y));
		min_y = min(min(uneven_y));
		uneven_y = (uneven_y - min_y). / (max_y - min_y);
	end*/
	__device__ float lumalaws2(int col, int row, int width, int height, int distance)
	{
		__shared__ float tmp[3];
		tmp[0] = 1.f / (distance*distance); // max_y
		tmp[1] = (width / 2.f)*(width / 2.f) + (height / 2.f)*(height / 2.f); // radius_square_max
		tmp[2] = cosf(atan2f(sqrtf(tmp[1]), distance)) / (tmp[1] + distance * distance); // min_y

		float radius_square = (col - width / 2)*(col - width / 2) + (row - height / 2)*(row - height / 2);
		float theta = atan2f(sqrtf(radius_square), distance);
		float uneven_y = cosf(theta) / (radius_square + distance * distance);

		// Projecting the luminance value to [0,1]
		uneven_y = (uneven_y - tmp[2]) / (tmp[0] - tmp[2]);
		return uneven_y;
	}

	__global__ void createUnevenY(int distance, cv::cuda::PtrStepSz<float> uneven_y)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int col = thread_id % uneven_y.cols;
		int row = thread_id / uneven_y.cols;

		if (col < uneven_y.cols && row < uneven_y.rows)
		{
			uneven_y(row, col) = lumalaws2(col, row, uneven_y.cols, uneven_y.rows, distance);
		}
	}

	__global__ void calcUnevenY(float *Y, int width, int height, int distance)
	{
		int thread_id = _get_threadId_grid2D_block1D();

		int row = thread_id / width;
		int col = thread_id % width;
		if (row < width && col < height)
		{
			float radius_square = (col - width / 2.f) * (col - width / 2.f) + (row - height / 2.f) * (row - height / 2.f);
			float theta = atan2f(sqrtf(radius_square), distance);
			float L_square = radius_square + distance * distance;
			Y[thread_id] = 1 * cosf(theta) / L_square;
		}
	}

	__global__ void reduceUevenY_UUC3(cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSz<uchar3> dst, int degree = 2, int distance = 1000)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int i = thread_id % src.cols;
		int j = thread_id / src.cols;

		if (i < src.cols && j < src.rows)
		{
			float max_y = 1.f / (distance*distance);
			float radius_square_max = (src.cols / 2.f)*(src.cols / 2.f) + (src.rows / 2.f)*(src.rows / 2.f);
			float min_y = cosf(atan2f(sqrtf(radius_square_max), distance)) / (radius_square_max + distance * distance);

			float y = 0.2126*src(j, i).z / 255.f + 0.7152*src(j, i).y / 255.f + 0.0722 *src(j, i).x / 255.f;
			float u = -0.09991*src(j, i).z / 255.f - 0.33609*src(j, i).y / 255.f + 0.436 *src(j, i).x / 255.f;
			float v = 0.615*src(j, i).z / 255.f - 0.55861*src(j, i).y / 255.f - 0.05639 *src(j, i).x / 255.f;

			float radius_square = (i - src.cols / 2)*(i - src.cols / 2) + (j - src.rows / 2)*(j - src.rows / 2);
			float theta = atan2f(sqrtf(radius_square), distance);
			float uneven_y = cosf(theta) / (radius_square + distance * distance);
			uneven_y = (uneven_y - min_y) / (max_y - min_y);

			float y_new = ((1.f - degree)*uneven_y + degree) * y;
			y_new = y_new > 1.f ? 1.f : y_new;

			float R = (1.f * y_new + 1.28033*v) * 255.f;
			float G = (1.f * y_new - 0.21482*u - 0.38059*v) * 255.f;
			float B = (1.f * y_new + 2.12798*u) * 255.f;
			dst(j, i).x = uchar(B > 255 ? 255 : B);
			dst(j, i).y = uchar(G > 255 ? 255 : G);
			dst(j, i).z = uchar(R > 255 ? 255 : R);
		}
	}
	__global__ void reduceUevenY_FFC1(cv::cuda::PtrStepSz<float1> src, cv::cuda::PtrStepSz<float1> dst, int degree = 2, int distance = 1000)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int i = thread_id % src.cols;
		int j = thread_id / src.cols;

		if (i < src.cols && j < src.rows)
		{
			float max_y = 1.f / (distance*distance);
			float radius_square_max = (src.cols / 2.f)*(src.cols / 2.f) + (src.rows / 2.f)*(src.rows / 2.f);
			float min_y = cosf(atan2f(sqrtf(radius_square_max), distance)) / (radius_square_max + distance * distance);

			float radius_square = (i - src.cols / 2)*(i - src.cols / 2) + (j - src.rows / 2)*(j - src.rows / 2);
			float theta = atan2f(sqrtf(radius_square), distance);
			float uneven_y = cosf(theta) / (radius_square + distance * distance);
			uneven_y = (uneven_y - min_y) / (max_y - min_y);

			float y_new = ((1.f - degree)*uneven_y + degree) * src(j, i).x;
			// NOTE: the luminance value should not be limited here, but should be limited before convert th RGB
			//y_new = y_new > 1.f ? 1.f : y_new;
			//sy_new = y_new < 0 ? 0 : y_new;
			dst(j, i).x = y_new;
		}
	}

	__global__ void reduceUnevenY(cv::cuda::PtrStepSz<uchar4> src, cv::cuda::PtrStepSz<uchar3> hsv, cv::cuda::PtrStepSz<float> uneven_y, float magnify, float magniy0)
	{
		int thread_id = _get_threadId_grid2D_block1D();
		int col = thread_id % uneven_y.cols;
		int row = thread_id / uneven_y.cols;

		if (col < uneven_y.cols && row < uneven_y.rows)
		{
			float p = magnify * (1 - uneven_y(row, col)) * float(hsv(row, col).y) / 255.f;

			src(row, col).x = (uchar)MIN(float(src(row, col).x)*(magniy0 + p), 255);
			src(row, col).y = (uchar)MIN(float(src(row, col).y)*(magniy0 + p), 255);
			src(row, col).z = (uchar)MIN(float(src(row, col).z)*(magniy0 + p), 255);
		}
	}
}

GPU_ALGO_BEGIN
void createUnevenY(int width, int height, int distance, cv::cuda::GpuMat & uneven_y, hipStream_t stream)
{
	if (uneven_y.empty())
		uneven_y = cv::cuda::GpuMat(height, width, CV_32FC1);

	::createUnevenY << < dim3(90, 90), 256, 0, stream >> > (distance, uneven_y);
}

void reduceUnevenY(cv::cuda::GpuMat & src, cv::cuda::GpuMat & uneven_y, std::array<cv::cuda::GpuMat, 2> &tmp, float magnify, float magnify0, hipStream_t stream)
{
	//cv::Mat t3; src.download(t3);
	//cv::Mat t1; uneven_y.download(t1);

	cv::cuda::cvtColor(src, tmp[0], cv::COLOR_BGRA2BGR);
	cv::cuda::cvtColor(tmp[0], tmp[1], cv::COLOR_BGR2HSV);

	::reduceUnevenY << < dim3(90, 90), 256, 0, stream >> > (src, tmp[1], uneven_y, magnify, magnify0);

	//cv::Mat t2; src.download(t2);
}
GPU_ALGO_END